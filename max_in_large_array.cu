#include<stdio.h>
#include<hip/hip_runtime.h>
#define K 10

__global__ void dkernel(unsigned *arr, int N)
{
  int i;
   unsigned maxi = arr[threadIdx.x*K];
   for(i = threadIdx.x*K ;i< (threadIdx.x*K+K) && i< N ;i++)
   {
    if (arr[i] > maxi)
    {
      maxi = arr[i];
    }
   }
   arr[threadIdx.x] = maxi;
}

int main()
{
  int N = 100;
  unsigned max2;
  unsigned *arr, *arrh;
  hipMalloc(&arr,N*sizeof(unsigned));
  arrh=(unsigned*)malloc(N*sizeof(unsigned));

  for(int i=0;i<N;i++){
    arrh[i]=i;
  }
  hipMemcpy(arr,arrh,N*sizeof(unsigned),hipMemcpyHostToDevice);
  dkernel<<< 1,N/K >>>(arr,N);
  dkernel<<< 1,K >>>(arr,K);
  
  hipMemcpy(&max2,&arr[0],sizeof(unsigned),hipMemcpyDeviceToHost);
  printf("max is %d \n",max2);
  return 0;
}
