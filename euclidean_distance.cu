#include<stdio.h>
#include<hip/hip_runtime.h>
#include<math.h>

  struct point
  {
    int a;
    int b ;
  } ;

  __global__ void dkernel(struct point *arrayOfPoints_gpu,int N,float *arr)
    {
     unsigned idx = blockIdx.x*blockDim.x+threadIdx.x;
      float x1 = arrayOfPoints_gpu[blockIdx.x].a;
      float y1 = arrayOfPoints_gpu[blockIdx.x].b;
      float x2 = arrayOfPoints_gpu[threadIdx.x].a;
      float y2 = arrayOfPoints_gpu[threadIdx.x].b;
      
      if (blockIdx.x < threadIdx.x)
         arr[idx]= sqrtf((x1-x2)*(x1-x2)+(y1-y2)*(y1-y2));                
    }

 int main()
 {
    int N=5;
    float *arr;
    float *arrh;
    arrh = (float*)malloc(N*N*sizeof(float));
    hipMalloc(&arr,N*N*sizeof(float));
    struct point arrayOfPoints[N];

    for(int i=0;i<N;i++)
    {
       struct point temp;
       temp.a = i;
       temp.b =i+3;
       arrayOfPoints[i] = temp;
    }
    for(int i=0;i<N;i++){
      printf("a is %d and b is %d \n",arrayOfPoints[i].a,arrayOfPoints[i].b);
    }
    struct point *arrayOfPoints_gpu;
    hipMalloc(&arrayOfPoints_gpu,N*sizeof(struct point));
    hipMemcpy(arrayOfPoints_gpu,arrayOfPoints,N*sizeof(struct point),hipMemcpyHostToDevice);
    dkernel <<< N,N >>>(arrayOfPoints_gpu,N,arr);
    hipMemcpy(arrh,arr,N*(N-1)/2*sizeof(float),hipMemcpyDeviceToHost);

    for(int i=0;i<N*(N-1)/2;i++)
    {
        printf("%f ",arrh[i]);
    }
    return 0;   
 }
