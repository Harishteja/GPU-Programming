#include<stdio.h>
#include<hip/hip_runtime.h>
//Encrypt: each character c becomes c+1. z becomes a.
__global__ void dkernel(char *str,int length)
{
   unsigned id = threadIdx.x ;
   if (id < length)
   {
    if (str[id]=='z')
    {
       str[id]='a';
    }
    else
     str[id] = str[id] + 1;
   }

}
// Encrypt: each ith character c becomes c+i.
__global__ void dkernel2(char *str,int length)
{
   unsigned id = threadIdx.x ;
   if (id < length)
   {  
    
    //str[id] = 'a' +  ( str[id]+id )%26
      str[id] = 'a' + (str[id]+id-'a')%26;
    //str[id] = str[id] + id%26;
   }

}
int main()
{
    char strh[100];  
    printf("Enter a string: ");
    scanf("%s", strh);  
    printf("You entered: %s\n", strh);
    int len = strlen(strh);

    char *str;
    hipMalloc(&str,(strlen(strh) + 1) * sizeof(char));
    hipMemcpy(str,strh,(strlen(strh) + 1) * sizeof(char),hipMemcpyHostToDevice);
    dkernel2<<< 1,10 >>>(str,len);
    hipDeviceSynchronize();
    hipMemcpy(strh,str,(strlen(strh) + 1) * sizeof(char),hipMemcpyDeviceToHost);
    printf("Encrypted string is %s \n",strh);
    return 0;     
}
