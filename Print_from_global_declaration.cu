#include<stdio.h>
#include<hip/hip_runtime.h>
char*msg="shriram\n";
#define N 10
__global__ void fun(char *da)
{
  printf("%s",da);
  // a[threadIdx.x] = threadIdx.x * threadIdx.x;
  // printf("%d\n", threadIdx.x*threadIdx.x);
}
int main()
{
  char *da;
  hipMalloc(&da,N*sizeof(char));
  hipMemcpy(da,msg,N*sizeof(char),hipMemcpyHostToDevice);
  fun<<< 1,N >>>(da);
  hipDeviceSynchronize();
  return 0;
}
