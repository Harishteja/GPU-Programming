
#include<stdio.h>
#include<hip/hip_runtime.h>
#define N 5
#define M 6
__global__ void dkernel(unsigned *matrix)
{
  unsigned id = threadIdx.x * blockDim.y + threadIdx.y;
  matrix[id] = id;
}
int main()
{
  dim3 block(M,N,1);
  int i,j;
  unsigned *matrix, *hmatrix;
  hipMalloc(&matrix,N*M*sizeof(unsigned));
  hipHostMalloc(&hmatrix,N*M*sizeof(unsigned), hipHostMallocDefault);
  // hmatrix = (unsigned *)malloc(N * M * sizeof(unsigned));
  dkernel<<<1,block>>>(matrix);
  hipMemcpy(hmatrix,matrix,N*M*sizeof(unsigned),hipMemcpyDeviceToHost);
  for (i=0;i<N;i++)
  {
    for (j=0;j<M;j++)
    {
      printf("%2d ",hmatrix[i*M+j]);
    }
    printf("\n");
  }
  hipFree(matrix);

}
