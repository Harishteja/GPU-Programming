#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
#define N 10
__global__ void dkernel(int *a,int *b,int *c)
{
  c[threadIdx.x] = a[threadIdx.x]*a[threadIdx.x]+ b[threadIdx.x]*b[threadIdx.x]*b[threadIdx.x]; 
}

  int main(){
    int i,a[N],b[N],c[N],*a_gpu[N],*b_gpu[N],*c_gpu[N];
    for (i =1;i<N;i++)
    {
       a[i] = i*i;
    }
        for (i =1;i<N;i++)
    {
       b[i] = i*i*i;
    }
    cudamalloc(&a_gpu,N*sizeof(int));
    cudamalloc(&b_gpu,N*sizeof(int));
    cudamalloc(&c_gpu,N*sizeof(int));
    hipMemcpy(a_gpu,a,N*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(b_gpu,b,N*sizeof(int),hipMemcpyHostToDevice);
    dkernel<<<1,N>>>(a_gpu,b_gpu,c_gpu);
    hipMemcpy(c,c_gpu,N*sizeof(int),hipMemcpyDeviceToHost);
    for (i=1;i<N;i++)
    {
      printf("%d",c[i]);
    }
  }
