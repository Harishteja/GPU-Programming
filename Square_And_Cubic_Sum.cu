#include<stdio.h>
#include<hip/hip_runtime.h>
#define N 10
__global__ void dkernel(int *a,int *b,int *c)
{
  c[threadIdx.x] = a[threadIdx.x]*a[threadIdx.x]+ b[threadIdx.x]*b[threadIdx.x]*b[threadIdx.x]; 
}

  int main(){
    int i,a[N],b[N],c[N],*a_gpu,*b_gpu,*c_gpu;
    for (i =0;i<N;i++)
    {
       a[i] = i*i;
    }
        for (i =0;i<N;i++)
    {
       b[i] = i*i*i;
    }
    hipMalloc(&a_gpu,N*sizeof(int));
    hipMalloc(&b_gpu,N*sizeof(int));
    hipMalloc(&c_gpu,N*sizeof(int));
    hipMemcpy(a_gpu,a,N*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(b_gpu,b,N*sizeof(int),hipMemcpyHostToDevice);
    dkernel<<<1,N>>>(a_gpu,b_gpu,c_gpu);
    hipMemcpy(c,c_gpu,N*sizeof(int),hipMemcpyDeviceToHost);
    for (i=0;i<N;i++)
    {
      printf("%d\n",c[i]);
    }
  }
