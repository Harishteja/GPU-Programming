#include<stdio.h>
#include<hip/hip_runtime.h>
#define N 32

 __global__ void dkernel(int *a){
       unsigned i = threadIdx.x;
        a[i] = 0; 
 }

int main(){
  int a[N],*arr,i;
     hipMalloc(&arr,sizeof(int)*N);
     dkernel<<< 1,N >>>(arr);
     hipMemcpy(a,arr,sizeof(int)*N,hipMemcpyDeviceToHost);
     for (i= 1;i<=N;i++)
         printf("%d",a[i]);
     return 0;   
}
