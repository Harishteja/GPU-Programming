#include<stdio.h>
#include<hip/hip_runtime.h>
#define N 64
__global__ void dkernel(unsigned *matrix,unsigned *result){
  unsigned idx = blockIdx.x*blockDim.x+threadIdx.x;
  //for (unsigned ii = 0; ii < N; ++ii) 
   unsigned ii = idx / N;
    unsigned jj = idx % N;
       //for (unsigned jj = 0; jj < N; ++jj)
       
         for (unsigned kk = 0; kk < N; ++kk) {
           result[ii * N + jj] +=
           matrix[ii * N + kk] * matrix[kk * N + jj];
       }
}
int main() {
  unsigned *matrix,*result;
  unsigned *hmatrix;
  int i,j;
  hipMalloc(&matrix,N*N*sizeof(unsigned));
  hipMalloc(&result,N*N*sizeof(unsigned));
  hmatrix = (unsigned*)malloc(N*N*sizeof(unsigned));
  for(i=0;i<N;i++){
    for(j=0;j<N;j++){
      hmatrix[i*N+j] = i*N+j;
    }
  }
  hipMemcpy(matrix,hmatrix,N*N*sizeof(unsigned),hipMemcpyHostToDevice);
  dkernel<<< N,N >>>(matrix,result);

  hipMemcpy(hmatrix,result,N*N*sizeof(unsigned),hipMemcpyDeviceToHost);
  for (i=0;i<N;i++){
    for(j=0;j<N;j++){
      printf("%d ",hmatrix[i*N+j]);
    }
    printf("\n");
  }
return 0;
}
