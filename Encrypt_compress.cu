#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void dkernel(unsigned *arr,int N)
{
  unsigned id = threadIdx.x;
  unsigned count = 1;
  //if (id<N)
  //{
   while (arr[id] = arr[id+1])
   {
    count +=1;
    id +=1;
   }
   arr[threadIdx.x] = count;
  }
  //}
int main()
{
  unsigned *arr;
  int N;
  int arrh[] = {0,0,0,1,1,0,1,0,0,0,1,0,0,0,1,1,1,1,0,1,1,1,0,1,0,0,0,1};
  N=sizeof(arrh)/sizeof(arrh[0]);

  hipMalloc(&arr,N*sizeof(unsigned)); 
  hipMemcpy(arr,arrh,sizeof(arrh),hipMemcpyHostToDevice);

  dkernel<<< 1,N >>>(arr,N);
  hipMemcpy(arrh,arr,N*sizeof(unsigned),hipMemcpyDeviceToHost);

  for (int i=0;i<N;i++)
  {
    printf("%d ",arrh[i]);
  }
  return 0;
}
