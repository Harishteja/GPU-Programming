#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void dkernel(unsigned *arr,unsigned *ans,int N,int* i)
{
  unsigned id = threadIdx.x;
  unsigned count = 1;
  if (id<N)
  {
   if (arr[id] == arr[id+1] && arr[id]!=arr[id-1])
   {
    while(arr[id] == arr[id+1])
    {count +=1;
    id +=1;}
    ans[i] = count;
    i++;
   }
   
  }
  }
int main()
{
  unsigned *arr,*dans,*di;
  int N,i=0;
  int arrh[] = {0,0,0,1,1,0,1,0,0,0,1,0,0,0,1,1,1,1,0,1,1,1,0,1,0,0,0,1};
 // int ans[] = {0,0,0,1,1,0,1,0,0,0,1,0,0,0,1,1,1,1,0,1,1,1,0,1,0,0,0,1};
  N=sizeof(arrh)/sizeof(arrh[0]);

  hipMalloc(&arr,N*sizeof(unsigned));
  hipMemcpy(arr,arrh,sizeof(arrh),hipMemcpyHostToDevice);
  hipMalloc(&dans,N*sizeof(unsigned));
  hipMalloc(&di,1*sizeof(unsigned));
  hipMemcpy(di,&i,sizeof(unsigned),hipMemcpyHostToDevice);
  //hipMemcpy(dans,ans,sizeof(ans),hipMemcpyHostToDevice);

  dkernel<<< 1,N >>>(arr,ans,N,di);
  hipMemcpy(arrh,ans,N*sizeof(unsigned),hipMemcpyDeviceToHost);

  for (int i=0;i<N;i++)
  {
    printf("%d ",arrh[i]);
  }
  return 0;
}
