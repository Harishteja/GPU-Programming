#include<stdio.h>
#include<hip/hip_runtime.h>
#define N 5
#define M 6
__global__ void dkernel(unsigned *matrix)
{
  unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
  matrix[id] = id;
}
int main()
{
  unsigned *matrix, *hmatrix;
  int i,j;
  hipMalloc(&matrix,N*M*sizeof(unsigned));
  hipHostMalloc(&hmatrix,N*M*sizeof(unsigned), hipHostMallocDefault);
  dkernel<<< N,M >>>(matrix);
  hipMemcpy(hmatrix,matrix,N*M*sizeof(unsigned),hipMemcpyDeviceToHost);
  for (i=0;i<N;i++)
  {
    for (j=0;j<M;j++)
    {
      printf("%2d ",hmatrix[i*M+j]);
    }
    printf("\n");
  }
  return 0;
}

