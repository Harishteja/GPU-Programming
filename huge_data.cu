#include<stdio.h>
#include<hip/hip_runtime.h>
#define BLOCKSIZE 1024
__global__ void dkernel(unsigned *vector,unsigned N){
  unsigned idx = blockIdx.x*blockDim.x+threadIdx.x;
  if (idx < N )
  vector[idx] = idx;
}

int main(  int n,char *str[]){
  int i;
  unsigned N = atoi(str[1]);
  unsigned *vector, *hvector;
  hipMalloc(&vector,N*sizeof(unsigned));
  hvector = (unsigned*)malloc(N*sizeof(unsigned));
  unsigned nblocks = ceil((float)N/BLOCKSIZE);
  printf("nblocks are %d \n",nblocks);

  dkernel <<<nblocks,BLOCKSIZE>>>(vector,N);
  hipMemcpy(hvector,vector,N*sizeof(unsigned),hipMemcpyDeviceToHost);
  for( i = 0;i<N;i++){
    printf("%d\n",hvector[i]);
  }
  hipFree(vector);
  free(hvector);
  return 0;
}
